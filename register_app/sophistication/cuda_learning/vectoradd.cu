#include "hip/hip_runtime.h"
#include "vectoradd.cuh"

__global__ void VectorAddKernel(ElementType* input_1, 
                                ElementType* input_2,
                                ElementType* output,
                                uint32_t ElementNum)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x; 
    if (idx < ElementNum){
        output[idx] = input_1[idx] + input_2[idx];
    }
}

void CallVectorAddKernelFunction(std::vector<ElementType> input_1, 
                                std::vector<ElementType> input_2, 
                                ElementType* output,
                                uint32_t ElementNum)
{   
    ElementType* d_input_1 = NULL;
    ElementType* d_input_2 = NULL;
    ElementType* d_output = NULL;
    hipMalloc(reinterpret_cast<void**>(&d_input_1), sizeof(ElementType) * ElementNum);
    hipMalloc(reinterpret_cast<void**>(&d_input_2), sizeof(ElementType) * ElementNum);
    hipMalloc(reinterpret_cast<void**>(&d_output), sizeof(ElementType) * ElementNum);
    
    hipMemcpy(d_input_1, input_1.data(), sizeof(ElementType) * ElementNum, hipMemcpyHostToDevice);
    hipMemcpy(d_input_2, input_2.data(), sizeof(ElementType) * ElementNum, hipMemcpyHostToDevice);
    hipMemcpy(d_output, output, sizeof(ElementType) * ElementNum, hipMemcpyHostToDevice);
 
    VectorAddKernel<<<1, 10>>>(d_input_1, d_input_2, d_output, ElementNum);

    hipMemcpy(output, d_output, sizeof(ElementType) * ElementNum, hipMemcpyDeviceToHost);
    hipFree(d_input_1);
    hipFree(d_input_2);
    hipFree(d_output);
}
